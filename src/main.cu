#include "hip/hip_runtime.h"
#include <image.h>
#include <scene.h>
#include <geometry.h>
#include <material.h>
#include <parser_v2.h>
#include <camera.h>
#include <mesh.h>
#include <spectrum.h>
#include <bsdf.h>

#include <lbvh.h>

#include "fluid_cut.h"

#define OUT_FILE "result.png"
#define BUNNY "/home/felpz/Documents/Bunny-LowPoly.stl"
#define LOW_POLY_DRAGON "/home/felpz/Documents/untitled1.obj"
#define DEMON "/home/felpz/Documents/demon.obj"

#define MAX2(a, b) (a) > (b) ? (a) : (b)
#define MAX3(a, b, c) MAX2(MAX2((a), (b)), (c))

inline void perlin_generate(glm::vec3 *p, int size){
    for(int i = 0; i < size; i += 1){
        float x = 2.0f*random_float() - 1.0f;
        float y = 2.0f*random_float() - 1.0f;
        float z = 2.0f*random_float() - 1.0f;
        p[i] = glm::normalize(glm::vec3(x,y,z));
    }
}

inline void permute(int *p, int size){
    for(int i = size-1; i > 0; i --){
        int target = int(random_float() * (i+1));
        int tmp = p[i];
        p[i] = p[target];
        p[target] = tmp;
    }
}

inline void perlin_generate_perm(int *p, int size){
    for(int i = 0; i < size; i += 1){
        p[i] = i;
    }
    permute(p, size);
}

inline __host__ 
void perlin_initialize(Perlin **perlin, int size){
    if(!(*perlin)){
        *perlin = (Perlin *)cudaAllocate(sizeof(Perlin));
        //TODO: Pack
        (*(perlin))->ranvec = (glm::vec3 *)cudaAllocate(size * sizeof(glm::vec3));
        (*(perlin))->permx  = (int *)cudaAllocate(size * sizeof(int));
        (*(perlin))->permy  = (int *)cudaAllocate(size * sizeof(int));
        (*(perlin))->permz  = (int *)cudaAllocate(size * sizeof(int));
    }
    
    (*perlin)->size = size;
    
    perlin_generate((*(perlin))->ranvec, size);
    perlin_generate_perm((*(perlin))->permx, size);
    perlin_generate_perm((*(perlin))->permy, size);
    perlin_generate_perm((*(perlin))->permz, size);
}


__global__ 
void init_random_states(Image *image){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int x = tid % image->width;
    int y = tid / image->width;
    if(x < image->width && y < image->height){
        hiprand_init(1234, tid, 0, &(image->states[tid]));
    }
}

__bidevice__ 
Spectrum get_sky(glm::vec3 rd){
    return Spectrum(0.f);
    Spectrum blue = Spectrum::FromRGB(0.123f, 0.34f, 0.9f);
    glm::vec3 s = glm::normalize(rd);
    float t = 1.2f * (glm::abs(s.y) + 1.0f);
    Spectrum v1 = (1.0f - t) * Spectrum(1.0f);
    Spectrum v2 = t * blue;
    if(v1.HasNaNs()){
        printf("Sky NaN v1 (%g) (%g %g %g)!\n", 1.0f - t, rd.x, rd.y, rd.z);
    }
    
    if(v2.HasNaNs()){
        printf("Sky NaN v2 (%g) (%g %g %g)!\n", t, rd.x, rd.y, rd.z);
    }
    
    return v1 + v2;
}

__device__
Spectrum trace_single(Ray &source, Scene *scene, hiprandState *state){
    hit_record record;
    if(!hit_scene(scene, source, 0.0001f, FLT_MAX, &record, state)){
        source.alive = 0;
        return get_sky(source.direction);
    }
    
    Material *material = &scene->material_table[record.mat_handle];
    glm::vec3 wo = -glm::normalize(source.direction);
    glm::vec3 wi;
    glm::vec2 u(random_float(state), random_float(state));
    float pdf = 1.0f;
    
    BxDFType sampled;
    
    BSDF bsdf(record.normal);
    
    material_sample(material, &record, &bsdf, scene);
    
    Spectrum s = BSDF_Sample_f(&bsdf, wo, &wi, u, &pdf, 
                               BSDF_ALL, &sampled);
    
    if(s.IsBlack() || ABS(pdf) < 0.0001){
        source.energy = Spectrum(0.f);
        return Spectrum(0.f);
    }
    
    source.origin = record.p;
    source.direction = glm::normalize(wi);
    source.energy *= s * glm::abs(glm::dot(source.direction, record.normal)) / pdf;
    return material->Le;
}

__device__ 
glm::vec3 get_color(Ray source, Scene *scene, hiprandState *state,
                    int max_bounces)
{
    Spectrum result(0.f);
    source.energy = Spectrum(1.f);
    source.alive = 1;
    for(int i = 0; i < max_bounces; i++){
        Spectrum Le = trace_single(source, scene, state);
        result += source.energy * Le;
        if(source.energy.IsBlack() || !Le.IsBlack()) break;
    }
    
    if(result.HasNaNs()){
        printf("NaN value on radiance!\n");
        result = Spectrum(1.f); //this makes the pixel visible
    }
    
    return result.ToRGB();
}

__global__ void RenderBatch(Image *image, Scene *scene, 
                            int samples, int total_samples)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int x = tid % image->width;
    int y = tid / image->width;
    
    if(x < image->width && y < image->height){
        Camera *camera = scene->camera;
        glm::vec3 color = image->pixels[tid];
        hiprandState *state = &image->states[tid];
        
        int max_bounces = 20;
        float invSamp = 1.0f / ((float)total_samples);
        for(int i = 0; i < samples; i += 1){
            float u1 = 2.0f * hiprand_uniform(state);
            float u2 = 2.0f * hiprand_uniform(state);
            float dx = (u1 < 1.0f) ? sqrt(u1) - 1.0f : 1.0f - sqrt(2.0f - u1);
            float dy = (u2 < 1.0f) ? sqrt(u2) - 1.0f : 1.0f - sqrt(2.0f - u2);
            
            float u = ((float)x + dx) / (float)image->width;
            float v = ((float)y + dy) / (float)image->height;
            Ray r = camera_get_ray(camera, u, v, state);
            glm::vec3 col = get_color(r, scene, state, max_bounces) * invSamp;
            color += col;
        }
        
        image->pixels[tid] = color;
    }
}

enum ToneMapAlgorithm{
    Reinhard,
    Exponential,
    NaughtyDog
};

inline __bidevice__
glm::vec3 ReinhardMap(glm::vec3 value, float exposure){
    (void)exposure;
    return (value / (value + 1.f));
}

inline __bidevice__
glm::vec3  NaughtyDogMap(glm::vec3 value, float exposure){
    float A = 0.15f;
    float B = 0.50f;
    float C = 0.10f;
    float D = 0.20f;
    float E = 0.02f;
    float F = 0.30f;
    float W = 11.2f;
    value *= exposure;
    value = ((value * (A*value+C*B)+D*E)/(value*(A*value+B)+D*F))-E/F;
    float white = ((W*(A*W+C*B)+D*E)/(W*(A*W+B)+D*F))-E/F;
    value /= white;
    return value;
}

inline __bidevice__
glm::vec3 ExponentialMap(glm::vec3 value, float exposure){
    return (glm::vec3(1.f) - glm::exp(-value * exposure));
}

__global__ void ToneMap(Image *image, int exposure, ToneMapAlgorithm algorithm){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int x = tid % image->width;
    int y = tid / image->width;
    
    if(x < image->width && y < image->height){
        glm::vec3 color = image->pixels[tid];
        glm::vec3 mapped = color;
        switch(algorithm){
            case ToneMapAlgorithm::Reinhard: {
                mapped = ReinhardMap(mapped, exposure);
            } break;
            
            case ToneMapAlgorithm::NaughtyDog:{
                mapped = NaughtyDogMap(mapped, exposure);
            } break;
            
            case ToneMapAlgorithm::Exponential:{
                mapped = ExponentialMap(color, exposure);
            } break;
            
            default:{
                printf("Unknown algorithm\n");
            }
        }
        
        image->pixels[tid] = mapped;
    }
}

void _render_scene(Scene *scene, Image *image, int &samples, int samplesPerBatch){
    size_t threads = 64;
    size_t blocks = (image->pixels_count + threads - 1)/threads;
    
    std::cout << "Generating per pixel RNG seed" << std::endl;
    init_random_states<<<blocks, threads>>>(image);
    cudaSynchronize();
    
    std::cout << "Path tracing... 0%" << std::endl;
    int rv = 0;
    int runs = samples / samplesPerBatch;
    int total = 0;
    for(int i = 0; i < runs; i += 1){
        RenderBatch<<<blocks, threads>>>(image, scene, samplesPerBatch,
                                         samples);
        rv = cudaSynchronize();
        if(rv != 0){
            printf("Cuda Failure. Aborting...\n");
            image_write(image, OUT_FILE, samples);
            exit(0);
        }
        float pct = 100.0f*(float(i + 1)/float(runs));
        std::cout.precision(4);
        std::cout << "Path tracing... " << pct << "%" << std::endl;
        total += samplesPerBatch;
    }
    
    samples = total;
    
    std::cout << std::endl;
    std::cout << "Tone Mapping..." << std::flush;
    ToneMap<<<blocks, threads>>>(image, 2.f, ToneMapAlgorithm::Exponential);
    rv = cudaSynchronize();
    if(rv != 0){
        std::cout << "Cuda Failure. Aborting..." << std::endl;
        image_write(image, OUT_FILE, samples);
        exit(0);
    }

    std::cout << "OK" << std::endl;
}

void render_scene(Scene *scene, Image *image, int samples, int samplesPerBatch){
    Timed("Rendering", _render_scene(scene, image, samples, samplesPerBatch));
}

int render_bsdf(){
    Image *image = image_new(800, 600);
    Scene *scene = scene_new();
    int samples = 500;
    int samplesPerBatch = 10;
    
    //glm::vec3 origin = glm::vec3(-25,20,15);
    //glm::vec3 target = glm::vec3(0,5,0);
    //glm::vec3 up = glm::vec3(0.f,1.0f,0.0f);
    
    //glm::vec3 origin = glm::vec3(0.5f,4.f,-7.f);
    glm::vec3 origin = glm::vec3(1.f);
    glm::vec3 target = glm::vec3(0,0,0.f);
//    glm::vec3 origin = glm::vec3(278, 278, -700);
    //glm::vec3 target = glm::vec3(278,278,0);
    glm::vec3 up = glm::vec3(0.f,1.0f,0.0f);
    
    scene->perlin = nullptr;
    perlin_initialize(&scene->perlin, 256);
    
    
    texture_handle solidRed = scene_add_texture_solid(scene, glm::vec3(0.65, 0.05, 0.05));
    texture_handle solidGray = scene_add_texture_solid(scene,glm::vec3(0.8f));
    texture_handle solidGreen = scene_add_texture_solid(scene,glm::vec3(0.12, 0.45, 0.15));
    
    texture_handle black = scene_add_texture_solid(scene, glm::vec3(0.f));
    texture_handle white = scene_add_texture_solid(scene, glm::vec3(1.));
    texture_handle kd1 = scene_add_texture_solid(scene, glm::vec3(0.8));
    texture_handle kd2 = scene_add_texture_solid(scene, glm::vec3(.4f));
    texture_handle sigma = scene_add_texture_solid(scene, glm::vec3(43.0f));
    
    texture_handle kd = scene_add_texture_solid(scene, glm::vec3(0.7,0.7,0.658));
    texture_handle ks = scene_add_texture_solid(scene, glm::vec3(0.7,0.7,0.658));
    texture_handle k2 = scene_add_texture_solid(scene, glm::vec3(0.4));

    texture_handle kdgreen = scene_add_texture_solid(scene, glm::vec3(0.6784313725490196, 1.f, 0.1843137254901961));
    
    
    Spectrum spec_emit = Spectrum::FromRGB(10.f,10.f,9.41f)*3.f;
    texture_handle milk = scene_add_texture_solid(scene, glm::vec3(1.f,1.f,0.941f));
    
    material_handle red = scene_add_matte_material(scene, solidRed, sigma);
    material_handle green = scene_add_matte_material(scene, solidGreen, sigma);
    material_handle emit = scene_add_matte_materialLe(scene, white, sigma, 
                                                      spec_emit);
    material_handle gray = scene_add_matte_material(scene, solidGray, sigma);
    
    
    material_handle bsdf1 = scene_add_matte_material(scene, white, sigma);
    material_handle bsdf2 = scene_add_plastic_material(scene, kd2, kd2, 0.03);
    material_handle bsdf3 = scene_add_plastic_material(scene, kd, ks, 0.1f);
    material_handle bsdf6 = scene_add_plastic_material(scene, k2, k2, 0.03f);
    material_handle bsdfPlastic = scene_add_plastic_material(scene, kdgreen,
                                                             kdgreen, 0.03f);
    material_handle bsdf4 = scene_add_glass_material(scene, black, kd1,
                                                     0.0, 0.0, 1.33f);
    
    material_handle glass2 = scene_add_glass_reflector_material(scene, milk, milk, 1.33f);
    
    material_handle mirror = scene_add_mirror_material(scene, kd1);
    
#if 0
    scene_add_rectangle_yz(scene, 0, 555, 0, 555, 555, green, 1);
    scene_add_rectangle_yz(scene, 0, 555, 0, 555, 0, red);
    scene_add_rectangle_xz(scene, 213, 343, 227, 332, 554, emit, 1, 1);
    scene_add_rectangle_xz(scene, 0, 555, 0, 555, 555, gray, 1);
    scene_add_rectangle_xz(scene, 0, 555, 0, 555, 0, gray);
    scene_add_rectangle_xy(scene, 0, 555, 0, 555, 555, gray, 1);
    
    scene_add_sphere(scene, glm::vec3(190, 90, 190), 90, glass2);
    scene_add_box(scene, glm::vec3(357.5, 165.0, 377.5), glm::vec3(165,330,165),
                  glm::vec3(0.0f,15.0f,0.0f), gray);
#endif

                                  
    glm::mat4 translate(1.0f);
    glm::mat4 scale(1.0f);
    glm::mat4 rot(1.0f);
    Transforms transform;
    
    /* Dragon mesh (reduced)
    translate = glm::translate(translate, glm::vec3(1.0f, 0.8f, -3.0f));
    scale = glm::scale(scale, glm::vec3(0.1f));
    origin = glm::vec3(1.f,4.f,-8.5f);
    rot = glm::rotate(rot, glm::radians(240.0f),
                      glm::vec3(0.0f,1.0f,0.0f));
                      
    transform.toWorld = translate * scale * rot;
    
    Mesh *mesh = load_mesh_obj(LOW_POLY_DRAGON, bsdfPlastic, transform);
    */
    
//    Mesh *mesh = load_mesh_obj(DEMON, bsdfPlastic, transform);
    
    //target = mesh->bvh->box.centroid;
    //transform.toWorld = glm::mat4(1.0f);
//    scene_add_mesh(scene, mesh, transform);
    
    
#if 0
    struct sph{
        float rad;
        glm::vec3 c;
    };
    
    std::vector<sph> pos;
    
    float boxlen = aabb_max_length(mesh->bvh->box)/2.f;
    
    auto test = [&](sph c){
        for(sph &s : pos){
            float d = glm::distance(s.c, c.c);
            float d2 = glm::distance(c.c, mesh->bvh->box.centroid);
            if(d < s.rad + c.rad || d2 < boxlen - c.rad) return false;
        }
        
        return true;
    };
    
    for(int i = 0; i < 80; i++){
        float rad = 0.f;
        glm::vec3 c(0.f);
        float g = 0.f;
        bool ok = false;
        sph pp;
        while(!ok){
            rad = 1.f + 2.f * random_float();
            c = glm::vec3(-20.f - 40.f * random_float(),
                          rad, -20.f + 40.f * random_float());
            pp.rad = rad;
            pp.c = c;
            ok = test(pp);
        }
        
        pos.push_back(pp);
        
        g = random_float();
        
        glm::vec3 c0(random_float(),random_float(),random_float());
        glm::vec3 c1(random_float(),random_float(),random_float());
        if(g < 0.5){
            float t = 0.1f * random_float();
            texture_handle hnd = scene_add_texture_solid(scene, c0);
            texture_handle hnd2 = scene_add_texture_solid(scene, c1);
            material_handle mat = scene_add_plastic_material(scene, hnd, hnd2, t);
            scene_add_sphere(scene, c, rad, mat);
        }else if(g < 0.85){
            g *= 10.f;
            texture_handle hnd = scene_add_texture_solid(scene, c0);
            texture_handle sig = scene_add_texture_solid(scene, glm::vec3(g));
            material_handle mat = scene_add_matte_material(scene, hnd, sig);
            scene_add_sphere(scene, c, rad, mat);
        }else if(g < 0.9){
            texture_handle hnd = scene_add_texture_solid(scene, c0);
            material_handle mat = scene_add_mirror_material(scene, hnd);
            scene_add_sphere(scene, c, rad, mat);
        }else{
            Spectrum Le = Spectrum::FromRGB(c0);
            texture_handle hnd = scene_add_texture_solid(scene, c0);
            material_handle emit2 = scene_add_matte_materialLe(scene, hnd, sigma, Le);
            scene_add_sphere(scene, c, rad, emit2);
        }
    }
    
    glm::vec3 v(0.f);
    for(sph &c : pos){
        v += c.c;
    }
    
    v /= (float)pos.size();
    
    target = v;
#endif
    
    /*
    samples = 100;
    samplesPerBatch = 10;
    origin = glm::vec3(5.f, 1.f, 5.f);
    target = glm::vec3(0.f,1.f,0.f);
    scene_add_sphere(scene, glm::vec3(0.0f, 1.f, 0.0f), 1.f, bsdf6);
    */
    scene_add_sphere(scene, glm::vec3(0.15f, 0.25f, 0.f), 0.25f, mirror);
    scene_add_sphere(scene, glm::vec3(-0.5f, 0.15f, 0.f), 0.15f, bsdfPlastic);
    scene_add_rectangle_xz(scene, -500, 500, -500, 500, 0.f, bsdf3);
    float w = 10.f;
    glm::vec3 v(0.f, 5.f, 0.f);
    
    //scene_add_rectangle_xz(scene, v.x-w, v.x+w, v.z-w, v.z+w, 10.f, emit);
    scene_add_sphere(scene, glm::vec3(4.f, 5.f, 2.f), 1.f, emit);
    
    
/*                          
    translate = glm::translate(glm::mat4(1.0f), glm::vec3(0.0f, 0.8f, .0f));
    scale = glm::scale(glm::mat4(1.0f), glm::vec3(0.02f));
    rot = glm::rotate(glm::mat4(1.0f), glm::radians(-90.0f),
                      glm::vec3(0.0f,1.0f,0.0f));
    rot = glm::rotate(rot, glm::radians(-90.0f),
                      glm::vec3(1.0f,0.0f,0.0f));
                      

    transform.toWorld = translate * scale * rot;
    Mesh *mesh2 = load_mesh_stl(BUNNY, red, transform);
    transform.toWorld = glm::mat4(1.0f);
    scene_add_mesh(scene, mesh2, transform);
*/    
    Timed("Building BVH", scene_build_done(scene));
    
    float aspect = (float)image->width / (float)image->height;
    float focus_dist = glm::length(origin - target);
    (void)focus_dist;
    
    scene->camera = camera_new(origin, target, up, 45, aspect);
    render_scene(scene, image, samples, samplesPerBatch);
    
    image_write(image, OUT_FILE, samples);
    image_free(image);
    return 0;
}

int change_val(int &s){
    s += 1;
    return s;
}

int main(int argc, char **argv){
    srand(time(0));
    (void)cudaInit();
    return render_bsdf();
}
