#include "hip/hip_runtime.h"
#include <image.h>
#include <scene.h>
#include <geometry.h>
#include <cuda_util.cuh>
#include <material.h>
#include <parser_v2.h>
#define OUT_FILE "result.png"

inline void perlin_generate(glm::vec3 *p, int size){
    for(int i = 0; i < size; i += 1){
        float x = 2.0f*random_float() - 1.0f;
        float y = 2.0f*random_float() - 1.0f;
        float z = 2.0f*random_float() - 1.0f;
        p[i] = glm::normalize(glm::vec3(x,y,z));
    }
}

inline void permute(int *p, int size){
    for(int i = size-1; i > 0; i --){
        int target = int(random_float() * (i+1));
        int tmp = p[i];
        p[i] = p[target];
        p[target] = tmp;
    }
}

inline void perlin_generate_perm(int *p, int size){
    for(int i = 0; i < size; i += 1){
        p[i] = i;
    }
    permute(p, size);
}

inline __host__ void perlin_initialize(Perlin **perlin, int size){
    if(!(*perlin)){
        CHECK(hipMallocManaged(perlin, sizeof(Perlin)));
        //TODO: Pack
        CHECK(hipMallocManaged(&(*(perlin))->ranvec, size * sizeof(glm::vec3)));
        CHECK(hipMallocManaged(&(*(perlin))->permx, size * sizeof(int)));
        CHECK(hipMallocManaged(&(*(perlin))->permy, size * sizeof(int)));
        CHECK(hipMallocManaged(&(*(perlin))->permz, size * sizeof(int)));
    }
    
    (*perlin)->size = size;
    
    perlin_generate((*(perlin))->ranvec, size);
    perlin_generate_perm((*(perlin))->permx, size);
    perlin_generate_perm((*(perlin))->permy, size);
    perlin_generate_perm((*(perlin))->permz, size);
}


__global__ void init_random_states(Image *image){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int x = tid % image->width;
    int y = tid / image->width;
    if(x < image->width && y < image->height){
        hiprand_init(1234, tid, 0, &(image->states[tid]));
    }
}

__host__ __device__ glm::vec3 get_sky(Ray r){
    return glm::vec3(0.0f);
    glm::vec3 dir = glm::normalize(r.direction);
    float t = 0.5f * (dir.y + 1.0f);
    return (1.0f - t) * glm::vec3(1.0f) + t*glm::vec3(0.5f, 0.7f, 1.0f);
}

__device__ glm::vec3 get_color(Ray source, Scene *scene, hiprandState *state, 
                               int max_bounces)
{
    glm::vec3 pixel(0.0f, 0.0f, 0.0f);
    glm::vec3 mask(1.0f, 1.0f, 1.0f);
    Ray r = source;
    Ray scattered;
    LightEval eval;
    Material *material = 0;
    for(int depth = 0; depth < max_bounces; depth += 1){
        hit_record record;
        /* Watch out for self intersection (0.001f) */
        if(!hit_scene(scene, r, 0.001f, FLT_MAX, 
                      &record, state))
        {
            pixel += mask * get_sky(r);
            break;
        }
        
        material = &scene->material_table[record.mat_handle];
        ray_sample_material(r, scene, material, &record, &eval, state);
        
        bool st = scatter(r, &record, scene, &eval, &scattered, material, state);
        
        pixel += mask * eval.emitted;
        
        mask *= eval.attenuation;
        
        if(!st){ break; }
        
        r = scattered;
    }
    return pixel;
}

__global__ void RenderBatch(Image *image, Scene *scene, 
                            int samples, int total_samples)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int x = tid % image->width;
    int y = tid / image->width;
    
    if(x < image->width && y < image->height){
        Camera *camera = scene->camera;
        glm::vec3 color = image->pixels[tid];
        hiprandState *state = &image->states[tid];
        
        int max_bounces = 5;
        
        for(int i = 0; i < samples; i += 1){
            float u1 = 2.0f * hiprand_uniform(state);
            float u2 = 2.0f * hiprand_uniform(state);
            float dx = (u1 < 1.0f) ? sqrt(u1) - 1.0f : 1.0f - sqrt(2.0f - u1);
            float dy = (u2 < 1.0f) ? sqrt(u2) - 1.0f : 1.0f - sqrt(2.0f - u2);
            
            float u = ((float)x + dx) / (float)image->width;
            float v = ((float)y + dy) / (float)image->height;
            Ray r = camera_get_ray(camera, u, v, state);
            color += get_color(r, scene, state, max_bounces) / (float)total_samples;
        }
        
        image->pixels[tid] = color;
    }
}

void _render_scene(Scene *scene, Image *image, int samples, int samplesPerBatch){
    size_t threads = 64;
    size_t blocks = (image->pixels_count + threads - 1)/threads;
    
    std::cout << "Generating per pixel RNG seed" << std::endl;
    init_random_states<<<blocks, threads>>>(image);
    cudaSynchronize();
    
    std::cout << "Path tracing..." << std::endl;
    
    int runs = samples / samplesPerBatch;
    for(int i = 0; i < runs; i += 1){
        RenderBatch<<<blocks, threads>>>(image, scene, samplesPerBatch,
                                         samples);
        cudaSynchronize();
        float pct = 100.0f*(float(i + 1)/float(runs));
        std::cout.precision(4);
        std::cout << "\r" << pct << "%    " << std::flush;
    }
    
    std::cout << std::endl;
}

void render_scene(Scene *scene, Image *image, int samples, int samplesPerBatch){
    Timed("Rendering", _render_scene(scene, image, samples, samplesPerBatch));
}

int render_fluid_scene(const char *path){
    Image *image = image_new(600, 400);
    Scene *scene = scene_new();
    Parser_v2 *parser = Parser_v2_new("vs");
    Timed("Reading particles", Parser_v2_load_single_file(parser, path));
    float radius = 0.012f;
    size_t n = 0;
    size_t bo = 0;
    
    glm::vec3 *particles = Parser_v2_get_raw_vector_ptr(parser, 0, 0, &n);
    float *boundary = Parser_v2_get_raw_scalar_ptr(parser, 0, 0, &n);
    
    for(size_t k = 0; k < n; k += 1){
        bo += boundary[k] ? 1 : 0;
    }
    
    std::cout << "Boundary " << bo << std::endl;
    scene->perlin = nullptr;
    perlin_initialize(&scene->perlin, 256);
    
    /* Build texture, all colors only */
    texture_handle tex_part = scene_add_texture_solid(scene, 
                                                      glm::vec3(0.98, 0.1, 0.2));
    texture_handle ground_tex = scene_add_texture_solid(scene,
                                                        glm::vec3(0.68));
    texture_handle glass_tex = scene_add_texture_solid(scene, glm::vec3(1.0f));
    
    /* Build materials */
    material_handle mat_part = scene_add_material_diffuse(scene, tex_part);
    material_handle mat_ground = scene_add_material_diffuse(scene, ground_tex);
    material_handle mat_glass = scene_add_material_dieletric(scene, 
                                                             glass_tex, 1.07f);
    
    //ground is giant sphere
    scene_add_sphere(scene, glm::vec3(0.0f, -1000.5f, -1.0f), 1000.0f, mat_ground);
    
    //add all particles
    for(size_t i = 0; i < n; i += 1){
        scene_add_sphere(scene, particles[i], radius, mat_part);
    }
    
    //container
    //scene_add_sphere(scene, glm::vec3(1.0f), 1.0f+2.0f*radius, mat_glass);
    Timed("Building BVH", scene_build_done(scene));
    
    //set camera stuff
    float aspect = (float)image->width / (float)image->height;
    glm::vec3 origin = glm::vec3(1.0f, 3.0f, 3.5f);
    glm::vec3 target = glm::vec3(1.0f);
    glm::vec3 up = glm::vec3(0.f,1.0f,0.0f);
    
    //in case you want focus, I don't use it
    float focus_dist = glm::length(origin - target);
    (void)focus_dist;
    
    //scene->camera = camera_new(origin, target, up, 45, aspect, 2.0f, focus_dist);
    scene->camera = camera_new(origin, target, up, 45, aspect);
    
    //define samples to run per pixel and per pixel per run
    int samples = 100;
    int samplesPerBatch = 10;
    
    //start path tracer
    render_scene(scene, image, samples, samplesPerBatch);
    
    image_write(image, OUT_FILE);
    image_free(image);
    return 0;
}

int render_cornell(){
    Image *image = image_new(800, 600);
    Scene *scene = scene_new();
    scene->perlin = nullptr;
    perlin_initialize(&scene->perlin, 256);
    texture_handle solidRed = scene_add_texture_solid(scene, glm::vec3(0.65, 0.05, 0.05));
    texture_handle solidGray = scene_add_texture_solid(scene,glm::vec3(0.73, 0.73, 0.73));
    texture_handle solidGreen = scene_add_texture_solid(scene,glm::vec3(0.12, 0.45, 0.15));
    texture_handle solidWhite = scene_add_texture_solid(scene,glm::vec3(10.0f));
    
    texture_handle imageTex = scene_add_texture_image(scene, "forest.png");
    texture_handle gridTex = scene_add_texture_image(scene, "forest_grid.png");
    texture_handle white = scene_add_texture_solid(scene, glm::vec3(1.0f));
    material_handle red = scene_add_material_diffuse(scene, solidRed);
    material_handle green = scene_add_material_diffuse(scene, solidGreen);
    material_handle emit = scene_add_material_emitter(scene, solidWhite);
    material_handle gray = scene_add_material_diffuse(scene, solidGray);
    
    material_handle imageMat = scene_add_material_diffuse(scene, imageTex);
    material_handle gridMat = scene_add_material_diffuse(scene, gridTex);
    material_handle glass = scene_add_material_dieletric(scene, white, 1.07f);
    
    scene_add_rectangle_yz(scene, 0, 555, 0, 555, 555, gridMat, 1);
    scene_add_rectangle_yz(scene, 0, 555, 0, 555, 0, gridMat);
    
    scene_add_rectangle_xz(scene, 213, 343, 227, 332, 554, emit);
    scene_add_rectangle_xz(scene, 0, 555, 0, 555, 555, gray, 1);
    scene_add_rectangle_xz(scene, 0, 555, 0, 555, 0, gray);
    scene_add_rectangle_xy(scene, 0, 555, 0, 555, 555, imageMat, 1);
    
    scene_add_sphere(scene, glm::vec3(277.0, 120.5, 277.0), -70.0f, glass);
    
    Timed("Building BVH", scene_build_done(scene));
    
    float aspect = (float)image->width / (float)image->height;
    glm::vec3 origin = glm::vec3(278, 278, -500);
    glm::vec3 target = glm::vec3(278,278,0);
    glm::vec3 up = glm::vec3(0.f,1.0f,0.0f);
    
    float focus_dist = glm::length(origin - target);
    (void)focus_dist;
    
    //scene->camera = camera_new(origin, target, up, 45, aspect, 2.0f, focus_dist);
    scene->camera = camera_new(origin, target, up, 43, aspect);
    int samples = 1000;
    int samplesPerBatch = 100;
    
    render_scene(scene, image, samples, samplesPerBatch);
    
    image_write(image, OUT_FILE);
    image_free(image);
    return 0;
}

int render_cornell2(){
    Image *image = image_new(800, 600);
    Scene *scene = scene_new();
    scene->perlin = nullptr;
    
    perlin_initialize(&scene->perlin, 256);
    texture_handle solidRed = scene_add_texture_solid(scene, glm::vec3(0.65, 0.05, 0.05));
    texture_handle solidGray = scene_add_texture_solid(scene,glm::vec3(0.73, 0.73, 0.73));
    texture_handle solidGreen = scene_add_texture_solid(scene,glm::vec3(0.12, 0.45, 0.15));
    texture_handle solidWhite = scene_add_texture_solid(scene,glm::vec3(10.0f));
    texture_handle white = scene_add_texture_solid(scene, glm::vec3(1.0f));
    
    material_handle red = scene_add_material_diffuse(scene, solidRed);
    material_handle green = scene_add_material_diffuse(scene, solidGreen);
    material_handle emit = scene_add_material_emitter(scene, solidWhite);
    material_handle gray = scene_add_material_diffuse(scene, solidGray);
    
    texture_handle solidMet = scene_add_texture_solid(scene, 
                                                      glm::vec3(0.8, 0.6, 0.2));
    material_handle metal1 = scene_add_material_metal(scene, solidMet, 1.0f);
    material_handle glass = scene_add_material_dieletric(scene, white, 1.5f);
    
    scene_add_rectangle_yz(scene, 0, 555, 0, 555, 555, green, 1);
    scene_add_rectangle_yz(scene, 0, 555, 0, 555, 0, red);
    scene_add_rectangle_xz(scene, 213, 343, 227, 332, 554, emit, 1);
    scene_add_rectangle_xz(scene, 0, 555, 0, 555, 555, gray, 1);
    scene_add_rectangle_xz(scene, 0, 555, 0, 555, 0, gray);
    scene_add_rectangle_xy(scene, 0, 555, 0, 555, 555, gray, 1);
    
    //scene_add_box(scene, glm::vec3(192.5, 82.5, 147.5), glm::vec3(165,165,165),
    //glm::vec3(0.0f, -18.0f,0.0f), gray);
    
    scene_add_sphere(scene, glm::vec3(190, 90, 190), 90, glass);
    scene_add_box(scene, glm::vec3(357.5, 165.0, 377.5), glm::vec3(165,330,165),
                  glm::vec3(0.0f,15.0f,0.0f), gray);
    
    Timed("Building BVH", scene_build_done(scene));
    
    float aspect = (float)image->width / (float)image->height;
    glm::vec3 origin = glm::vec3(278, 278, -700);
    glm::vec3 target = glm::vec3(278,278,0);
    glm::vec3 up = glm::vec3(0.f,1.0f,0.0f);
    
    float focus_dist = glm::length(origin - target);
    (void)focus_dist;
    
    //scene->camera = camera_new(origin, target, up, 45, aspect, 2.0f, focus_dist);
    scene->camera = camera_new(origin, target, up, 43, aspect);
    int samples = 30000;
    int samplesPerBatch = 100;
    
    render_scene(scene, image, samples, samplesPerBatch);
    
    image_write(image, OUT_FILE);
    image_free(image);
    return 0;
}


int render_scene(){
    Object ball, air;
    Image *image = image_new(1366, 720);
    Scene *scene = scene_new();
    scene->perlin = nullptr;
    perlin_initialize(&scene->perlin, 256);
    
    int nb = 20;
    texture_handle solidWhite = scene_add_texture_solid(scene, glm::vec3(0.73));
    texture_handle solidGreen = scene_add_texture_solid(scene, glm::vec3(0.48, 0.83,
                                                                         0.53));
    texture_handle brightWhite = scene_add_texture_solid(scene, glm::vec3(7.0f));
    texture_handle solidBright = scene_add_texture_solid(scene, glm::vec3(0.8,0.8,0.9));
    
    texture_handle pertext = scene_add_texture_noise(scene, NOISE_TRILINEAR,
                                                     glm::vec3(1.0f));
    
    material_handle ground = scene_add_material_diffuse(scene, solidGreen);
    material_handle brightLight = scene_add_material_emitter(scene, brightWhite);
    material_handle glass = scene_add_material_dieletric(scene, solidWhite, 1.5f);
    material_handle iso = scene_add_material_isotropic(scene, glm::vec3(0.2,0.4,
                                                                        0.9));
    material_handle iso2 = scene_add_material_isotropic(scene, glm::vec3(0.9,0.4,
                                                                         0.1));
    
    
    material_handle white = scene_add_material_diffuse(scene, scene->white_texture);
    material_handle noisemat = scene_add_material_diffuse(scene, pertext);
    material_handle wwh = scene_add_material_diffuse(scene, solidWhite);
    
    
    for(int i = 0; i < nb; i += 1){
        for(int j = 0; j < nb; j += 1){
            float w = 100.0f;
            float x0 = -1000.0f + i * w;
            float z0 = -1000.0f + j * w;
            float y0 = 0.0f;
            
            float x1 = x0 + w;
            float y1 = 100.0f * (random_float() + 0.01f);
            float z1 = z0 + w;
            
            glm::vec3 p = glm::vec3((x0+x1)/2.0f, (y0+y1)/2.0f, (z0+z1)/2.0f); //pos
            glm::vec3 s = glm::vec3((x1-x0), (y0-y1), (z1-z0)); //scale
            glm::vec3 r = glm::vec3(0.0f); //rotation
            scene_add_box(scene, p, s, r, ground);
        }
    }
    
    
    scene_add_rectangle_xz(scene, 123, 423, 147, 412, 554, brightLight);
    scene_add_sphere(scene, glm::vec3(160, 170, 80), 70, glass);
    
    scene_add_sphere(scene, glm::vec3(360, 150, 145), 70, glass);
    ball = scene_add_sphere(scene, glm::vec3(360, 150, 145), 70, glass);
    scene_add_medium(scene, ball, 0.2f, iso);
    
    air = scene_add_sphere(scene, glm::vec3(0.0f), 5000.0f, glass);
    scene_add_medium(scene, air, 0.0001f, white);
    
    
    Object data;
    glm::vec3 boxp = glm::vec3(-70, 170, 175);
    glm::vec3 boxsize = glm::vec3(100.0f) * 0.5f ;
    
    scene_add_sphere(scene, boxp, boxsize.x, glass);
    data = scene_add_sphere(scene, boxp, boxsize.x, glass);
    //scene_add_box(scene, boxp, boxsize, glm::vec3(0.0f), glass);
    //data = scene_add_box(scene, boxp, boxsize, glm::vec3(0.0f), glass);
    
    scene_add_medium(scene, data, 0.2f, iso2);
    
    scene_add_sphere(scene, glm::vec3(250,280,300), 80, noisemat);
    
    int ns = 1000;
    for(int j = 0; j < ns; j += 1){
        float x = 165.0f * random_float();
        float y = 165.0f * random_float();
        float z = 165.0f * random_float();
        glm::vec3 c(x, y, z);
        scene_add_sphere(scene, c + glm::vec3(-100.0f, 270.0f, 395.0f), 10.0f, wwh);
    }
    
    Timed("Building BVH", scene_build_done(scene));
    
    float aspect = (float)image->width / (float)image->height;
    glm::vec3 origin = glm::vec3(478, 278, -600);
    glm::vec3 target = glm::vec3(278,278,0);
    glm::vec3 up = glm::vec3(0.f,1.0f,0.0f);
    
    scene->camera = camera_new(origin, target, up, 40, aspect);
    int samples = 30000;
    int samplesPerBatch = 100;
    
    render_scene(scene, image, samples, samplesPerBatch);
    
    image_write(image, OUT_FILE);
    image_free(image);
    return 0;
}


int main(int argc, char **argv){
    (void)cudaInit();
    
    //return render_scene();
    return render_cornell2();
    //return render_fluid_scene("/home/felpz/OUT_PART_SimplexSphere2_60.txt");
    //return render_fluid_scene("/home/felpz/OUT_PART_3DRun_10.txt");
    
    
    Image *image = image_new(800, 600);
    Scene *scene = scene_new();
    scene->perlin = nullptr;
    perlin_initialize(&scene->perlin, 256);
    
    texture_handle imageTexture = scene_add_texture_image(scene, "earthmap.jpg");
    
    texture_handle noiseTexture = scene_add_texture_noise(scene, NOISE_TRILINEAR,
                                                          glm::vec3(1.0f));
    
    texture_handle solidGreen = scene_add_texture_solid(scene,
                                                        glm::vec3(0.1,0.6,0.1));
    texture_handle solidRed = scene_add_texture_solid(scene, glm::vec3(0.9,0.1,0.1));
    
    texture_handle solidBlue = scene_add_texture_solid(scene, 
                                                       glm::vec3(0.1, 0.2, 0.7));
    
    texture_handle solidYellow = scene_add_texture_solid(scene,
                                                         glm::vec3(0.8, 0.8, 0.0));
    
    texture_handle solidMet = scene_add_texture_solid(scene, 
                                                      glm::vec3(0.8, 0.6, 0.2));
    
    texture_handle solidMet2 = scene_add_texture_solid(scene, 
                                                       glm::vec3(0.8, 0.8, 0.8));
    
    texture_handle solidWhite = scene_add_texture_solid(scene, glm::vec3(1.0f));
    
    texture_handle checker = scene_add_texture_checker(scene, 
                                                       solidWhite, solidGreen);
    
    material_handle diffuse1 = scene_add_material_diffuse(scene, solidBlue);
    
    material_handle diffuse2 = scene_add_material_diffuse(scene, checker);
    //material_handle diffuse2 = scene_add_material_diffuse(scene, noiseTexture);
    
    material_handle imageMat = scene_add_material_diffuse(scene, imageTexture);
    
    material_handle metal1 = scene_add_material_metal(scene, solidMet, 1.0f);
    
    material_handle metal2 = scene_add_material_metal(scene, solidMet2, 0.3f);
    
    material_handle glass = scene_add_material_dieletric(scene, solidWhite, 1.07f);
    
    material_handle emit = scene_add_material_emitter(scene, solidGreen);
    material_handle emit2 = scene_add_material_emitter(scene, solidRed);
    material_handle emit3 = scene_add_material_emitter(scene, solidBlue);
    material_handle emit4 = scene_add_material_emitter(scene, scene->white_texture);
    
    scene_add_sphere(scene, glm::vec3(0.0f, 0.0f, -1.0f), 0.5f, diffuse1);
    scene_add_sphere(scene, glm::vec3(0.0f, -1000.5f, -1.0f), 1000.0f, diffuse2);
    
    scene_add_sphere(scene, glm::vec3(1.0f+0.001f,0.0f,-1.0f), -0.45f, glass);
    scene_add_sphere(scene, glm::vec3(-1.0f-0.001f,0.0f,-1.0f), 0.5f, imageMat);
    
    scene_add_rectangle_yz(scene, -0.2f, 1.5f, -2.2f, 1.2f, -2.1f, emit);
    scene_add_rectangle_yz(scene, -0.2f, 1.5f, -2.2f, 1.2f, 2.1f, emit2);
    scene_add_rectangle_xy(scene, -1.7f, 1.7f, -0.2f, 1.5f, 1.2f, emit4);
    //scene_add_sphere(scene, glm::vec3(0.0f,2.3f,-1.0f), 0.35f, emit3);
    
    Timed("Building BVH", scene_build_done(scene));
    
    float aspect = (float)image->width / (float)image->height;
    glm::vec3 origin = glm::vec3(0.f,3.f,-7.0f);
    glm::vec3 target = glm::vec3(0.0f,0.0f,-1.0f);
    glm::vec3 up = glm::vec3(0.f,1.0f,0.0f);
    
    float focus_dist = glm::length(origin - target);
    (void)focus_dist;
    
    //scene->camera = camera_new(origin, target, up, 45, aspect, 2.0f, focus_dist);
    scene->camera = camera_new(origin, target, up, 45, aspect);
    int samples = 1000;
    int samplesPerBatch = 100;
    
    render_scene(scene, image, samples, samplesPerBatch);
    
    image_write(image, OUT_FILE);
    image_free(image);
    return 0;
}
